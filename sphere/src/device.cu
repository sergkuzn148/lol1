#include "hip/hip_runtime.h"
// device.cu -- GPU specific operations utilizing the CUDA API.
#include <iostream>
#include <fstream>
#include <string>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <iomanip>
#include <time.h>

#include "vector_arithmetic.h"  // for arbitrary prec. vectors
//#include <vector_functions.h> // for single prec. vectors
#include "thrust/device_ptr.h"
#include "thrust/sort.h"

#include "sphere.h"
#include "datatypes.h"
#include "utility.h"
#include "constants.cuh"
#include "debug.h"
#include "version.h"

#include "sorting.cuh"
#include "contactmodels.cuh"
#include "cohesion.cuh"
#include "contactsearch.cuh"
#include "integration.cuh"
#include "raytracer.cuh"
#include "navierstokes.cuh"
#include "darcy.cuh"

// Returns the number of cores per streaming multiprocessor, which is
// a function of the device compute capability
// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#compute-capabilities
int cudaCoresPerSM(int major, int minor)
{
    if (major == 1)
        return 8;
    else if (major == 2 && minor == 0)
        return 32;
    else if (major == 2 && minor == 1)
        return 48;
    else if (major == 3)
        return 192;
    else if (major == 4)
        return 128;
    else if (major == 5)
        return 128;
    else if (major == 6 && minor == 0)
        return 64;
    else if (major == 6 && minor == 1)
        return 128;
    else if (major == 6 && minor == 2)
        return 128;
    else if (major == 7)
        return 32;
    else
        printf("Error in cudaCoresPerSM Device compute capability value "
                "(%d.%d) not recognized.", major, minor);
    return -1;
}

// Wrapper function for initializing the CUDA components.
// Called from main.cpp
void DEM::initializeGPU(void)
{
    using std::cout; // stdout

    // Specify target device
    int cudadevice = 0;

    // Variables containing device properties
    hipDeviceProp_t prop;
    int deviceCount;
    int cudaDriverVersion;
    int cudaRuntimeVersion;

    checkForCudaErrors("Before initializing CUDA device");

    // Register number of devices
    hipGetDeviceCount(&deviceCount);
    ndevices = deviceCount; // store in DEM class

    if (deviceCount == 0) {
        std::cerr << "\nERROR: No CUDA-enabled devices availible. Bye."
            << std::endl;
        exit(EXIT_FAILURE);
    } else if (deviceCount == 1) {
        if (verbose == 1)
            cout << "  System contains 1 CUDA compatible device.\n";
    } else {
        if (verbose == 1)
            cout << "  System contains " << deviceCount
                << " CUDA compatible devices.\n";
    }

    // Loop through GPU's and choose the one with the most CUDA cores
    if (device == -1) {
        int ncudacores;
        int max_ncudacores = 0;
        for (int d=0; d<ndevices; d++) {
            hipGetDeviceProperties(&prop, d);
            hipDriverGetVersion(&cudaDriverVersion);
            hipRuntimeGetVersion(&cudaRuntimeVersion);

            ncudacores = prop.multiProcessorCount
                *cudaCoresPerSM(prop.major, prop.minor);
            if (ncudacores > max_ncudacores) {
                max_ncudacores = ncudacores;
                cudadevice = d;
            }

            if (verbose == 1) {
                cout << "  CUDA device ID: " << d << "\n";
                cout << "  - Name: " <<  prop.name << ", compute capability: " 
                     << prop.major << "." << prop.minor << ".\n";
                cout << "  - CUDA Driver version: " << cudaDriverVersion/1000 
                     << "." <<  cudaDriverVersion%100 
                     << ", runtime version " << cudaRuntimeVersion/1000 << "." 
                     << cudaRuntimeVersion%100 << std::endl;
            }
        }

        device = cudadevice; // store in DEM class
        if (verbose == 1) {
            cout << "  Using CUDA device ID " << device << " with "
                 << max_ncudacores << " cores." << std::endl;
        }

    } else {

        hipGetDeviceProperties(&prop, device);
        hipDriverGetVersion(&cudaDriverVersion);
        hipRuntimeGetVersion(&cudaRuntimeVersion);

        int ncudacores = prop.multiProcessorCount
            *cudaCoresPerSM(prop.major, prop.minor);

        if (verbose == 1) {
            cout << "  CUDA device ID: " << device << "\n";
            cout << "  - Name: " <<  prop.name << ", compute capability: " 
                 << prop.major << "." << prop.minor << ".\n";
            cout << "  - CUDA Driver version: " << cudaDriverVersion/1000 
                 << "." <<  cudaDriverVersion%100 
                 << ", runtime version " << cudaRuntimeVersion/1000 << "." 
                 << cudaRuntimeVersion%100
                 << "\n  - " << ncudacores << " CUDA cores" << std::endl;
        }
    }

    // The value of device is now 0 or larger
    hipSetDevice(device);

    checkForCudaErrors("While initializing CUDA device");
}

// Start timer for kernel profiling
void startTimer(hipEvent_t* kernel_tic)
{
    hipEventRecord(*kernel_tic);
}

// Stop timer for kernel profiling and time to function sum
void stopTimer(hipEvent_t *kernel_tic,
        hipEvent_t *kernel_toc,
        float *kernel_elapsed,
        double* sum)
{
    hipEventRecord(*kernel_toc, 0);
    hipEventSynchronize(*kernel_toc);
    hipEventElapsedTime(kernel_elapsed, *kernel_tic, *kernel_toc);
    *sum += *kernel_elapsed;
}

// Check values of parameters in constant memory
__global__ void checkConstantValues(int* dev_equal,
        Grid* dev_grid,
        Params* dev_params)
{
    // Values ok (0)
    *dev_equal = 0;

    // Compare values between global- and constant
    // memory structures
    if (dev_grid->origo[0] != devC_grid.origo[0])
        *dev_equal = 1;
    if (dev_grid->origo[1] != devC_grid.origo[1])
        *dev_equal = 2; // Not ok
    if (dev_grid->origo[2] != devC_grid.origo[2])
        *dev_equal = 3; // Not ok
    if (dev_grid->L[0] != devC_grid.L[0])
        *dev_equal = 4; // Not ok
    if (dev_grid->L[1] != devC_grid.L[1])
        *dev_equal = 5; // Not ok
    if (dev_grid->L[2] != devC_grid.L[2])
        *dev_equal = 6; // Not ok
    if (dev_grid->num[0] != devC_grid.num[0])
        *dev_equal = 7; // Not ok
    if (dev_grid->num[1] != devC_grid.num[1])
        *dev_equal = 8; // Not ok
    if (dev_grid->num[2] != devC_grid.num[2])
        *dev_equal = 9; // Not ok
    if (dev_grid->periodic != devC_grid.periodic)
        *dev_equal = 10; // Not ok

    if (dev_params->g[0] != devC_params.g[0])
        *dev_equal = 11; // Not ok
    if (dev_params->g[1] != devC_params.g[1])
        *dev_equal = 12; // Not ok
    if (dev_params->g[2] != devC_params.g[2])
        *dev_equal = 13; // Not ok
    if (dev_params->k_n != devC_params.k_n)
        *dev_equal = 14; // Not ok
    if (dev_params->k_t != devC_params.k_t)
        *dev_equal = 15; // Not ok
    if (dev_params->k_r != devC_params.k_r)
        *dev_equal = 16; // Not ok
    if (dev_params->gamma_n != devC_params.gamma_n)
        *dev_equal = 17; // Not ok
    if (dev_params->gamma_t != devC_params.gamma_t)
        *dev_equal = 18; // Not ok
    if (dev_params->gamma_r != devC_params.gamma_r)
        *dev_equal = 19; // Not ok
    if (dev_params->mu_s != devC_params.mu_s)
        *dev_equal = 20; // Not ok
    if (dev_params->mu_d != devC_params.mu_d)
        *dev_equal = 21; // Not ok
    if (dev_params->mu_r != devC_params.mu_r)
        *dev_equal = 22; // Not ok
    if (dev_params->rho != devC_params.rho)
        *dev_equal = 23; // Not ok
    if (dev_params->contactmodel != devC_params.contactmodel)
        *dev_equal = 24; // Not ok
    if (dev_params->kappa != devC_params.kappa)
        *dev_equal = 25; // Not ok
    if (dev_params->db != devC_params.db)
        *dev_equal = 26; // Not ok
    if (dev_params->V_b != devC_params.V_b)
        *dev_equal = 27; // Not ok
    if (dev_params->lambda_bar != devC_params.lambda_bar)
        *dev_equal = 28; // Not ok
    if (dev_params->nb0 != devC_params.nb0)
        *dev_equal = 29; // Not ok
    if (dev_params->E != devC_params.E)
        *dev_equal = 30; // Not ok
}

__global__ void checkParticlePositions(
    const Float4* __restrict__ dev_x)
{
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; // Thread id

    if (idx < devC_np) { // Condition prevents block size error
        Float4 x = dev_x[idx];

        // make sure grain doesn't have NaN or Inf position
        if (!isfinite(x.x) || !isfinite(x.y) || !isfinite(x.z)) {
            __syncthreads();
            printf("\nParticle %d has non-finite position: x = %f %f %f",
                    idx, x.x, x.y, x.z);
        }

        /*__syncthreads();
        printf("\nParticle %d: x = %f %f %f",
                idx, x.x, x.y, x.z);*/

        // check that the particle is inside of the simulation domain
        if (x.x < devC_grid.origo[0] ||
                x.y < devC_grid.origo[1] ||
                x.z < devC_grid.origo[2] ||
                x.x > devC_grid.L[0] ||
                x.y > devC_grid.L[1] ||
                x.z > devC_grid.L[2]) {
            __syncthreads();
            printf("\nParticle %d is outside the computational domain "
                    "(%f %f %f to %f %f %f): x = %f %f %f",
                    idx,
                    devC_grid.origo[0], devC_grid.origo[1], devC_grid.origo[2],
                    devC_grid.L[0], devC_grid.L[1], devC_grid.L[2],
                    x.x, x.y, x.z);
        }
    }
}


// Copy the constant data components to device memory,
// and check whether the values correspond to the 
// values in constant memory.
void DEM::checkConstantMemory()
{
    // Allocate space in global device memory
    Grid* dev_grid;
    Params* dev_params;
    hipMalloc((void**)&dev_grid, sizeof(Grid));
    hipMalloc((void**)&dev_params, sizeof(Params));

    // Copy structure data from host to global device memory
    hipMemcpy(dev_grid, &grid, sizeof(Grid), hipMemcpyHostToDevice);
    hipMemcpy(dev_params, &params, sizeof(Params), hipMemcpyHostToDevice);

    // Compare values between global and constant memory
    // structures on the device.
    int* equal = new int;  // The values are equal = 0, if not = 1
    *equal = 0;
    int* dev_equal;
    hipMalloc((void**)&dev_equal, sizeof(int));
    checkConstantValues<<<1,1>>>(dev_equal, dev_grid, dev_params);
    checkForCudaErrors("After constant memory check");

    // Copy result to host
    hipMemcpy(equal, dev_equal, sizeof(int), hipMemcpyDeviceToHost);

    // Free global device memory
    hipFree(dev_grid);
    hipFree(dev_params);
    hipFree(dev_equal);

    // Are the values equal?
    if (*equal != 0) {
        std::cerr << "Error! The values in constant memory do not "
            << "seem to be correct (" << *equal << ")." << std::endl;
        exit(1);
    } else {
        if (verbose == 1)
            std::cout << "  Constant values ok (" << *equal << ")."
                << std::endl;
    }
}

// Copy selected constant components to constant device memory.
void DEM::transferToConstantDeviceMemory()
{
    using std::cout;

    if (verbose == 1)
        cout << "  Transfering data to constant device memory:     ";

    /*for (int d=0; d<ndevices; d++) {
      hipSetDevice(d);*/
        hipMemcpyToSymbol(HIP_SYMBOL(devC_nd), &nd, sizeof(nd));
        hipMemcpyToSymbol(HIP_SYMBOL(devC_np), &np, sizeof(np));
        hipMemcpyToSymbol(HIP_SYMBOL(devC_nw), &walls.nw, sizeof(unsigned int));
        hipMemcpyToSymbol(HIP_SYMBOL(devC_nc), &NC, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(devC_dt), &time.dt, sizeof(Float));
        hipMemcpyToSymbol(HIP_SYMBOL(devC_grid), &grid, sizeof(Grid));
        hipMemcpyToSymbol(HIP_SYMBOL(devC_params), &params, sizeof(Params));
        /*}
          hipSetDevice(device);*/

    checkForCudaErrors("After transferring to device constant memory");

    if (verbose == 1)
        cout << "Done\n";

    // only for device with most CUDA cores
    checkConstantMemory();
}

__global__ void printWorldSize(Float4* dev_walls_nx)
{
    printf("\nL = %f, %f, %f\n",
            devC_grid.L[0], devC_grid.L[1], devC_grid.L[2]);
    printf("\ndev_walls_nx[0] = %f, %f, %f, %f\n",
            dev_walls_nx[0].x,
            dev_walls_nx[0].y,
            dev_walls_nx[0].z,
            dev_walls_nx[0].w);
}

void DEM::updateGridSize()
{
    //printf("\nDEM::updateGridSize() start\n");
    Float* Lz = new Float;

    // Get top wall position from dev_walls_nx[0].z
    hipMemcpy(Lz, &dev_walls_nx[0].w, sizeof(Float), hipMemcpyDeviceToHost);
    checkForCudaErrors("DEM::updateGridSize(): copying wall position");

    //printWorldSize<<<1,1>>>(dev_walls_nx);
    //hipDeviceSynchronize();
    //checkForCudaErrors("DEM::updateGridSize(): first printWorldSize");

    //printf("\nLz = %f\n", *Lz);

    // Write value to grid.L[2]
    grid.L[2] = *Lz;

    // Write value to devC_grid.L[2]
    //hipMemcpyToSymbol(HIP_SYMBOL(devC_grid.L[2]), &Lz, sizeof(Float)); 
    hipMemcpyToSymbol(HIP_SYMBOL(devC_grid), &grid, sizeof(Grid));

    checkForCudaErrors("DEM::updateGridSize(): write to devC_grid.L[2]");

    //printWorldSize<<<1,1>>>(dev_walls_nx);
    //hipDeviceSynchronize();
    //checkForCudaErrors("DEM::updateGridSize(): second printWorldSize");

    // check value only during debugging
    //checkConstantMemory();
}


// Allocate device memory for particle variables,
// tied to previously declared pointers in structures
void DEM::allocateGlobalDeviceMemory(void)
{
    // Particle memory size
    unsigned int memSizeF  = sizeof(Float) * np;
    unsigned int memSizeF4 = sizeof(Float4) * np;

    if (verbose == 1)
        std::cout << "  Allocating global device memory:                ";

    k.acc = new Float4[np];
    k.angacc = new Float4[np];
#pragma omp parallel for if(np>100)
    for (unsigned int i = 0; i<np; ++i) {
        k.acc[i] = MAKE_FLOAT4(0.0, 0.0, 0.0, 0.0);
        k.angacc[i] = MAKE_FLOAT4(0.0, 0.0, 0.0, 0.0);
    }

    // Kinematics arrays
    hipMalloc((void**)&dev_x, memSizeF4);
    hipMalloc((void**)&dev_xyzsum, memSizeF4);
    hipMalloc((void**)&dev_vel, memSizeF4);
    hipMalloc((void**)&dev_vel0, memSizeF4);
    hipMalloc((void**)&dev_acc, memSizeF4);
    hipMalloc((void**)&dev_force, memSizeF4);
    hipMalloc((void**)&dev_angpos, memSizeF4);
    hipMalloc((void**)&dev_angvel, memSizeF4);
    hipMalloc((void**)&dev_angvel0, memSizeF4);
    hipMalloc((void**)&dev_angacc, memSizeF4);
    hipMalloc((void**)&dev_torque, memSizeF4);

    // Particle contact bookkeeping arrays
    hipMalloc((void**)&dev_contacts,
               sizeof(unsigned int)*np*NC);
    hipMalloc((void**)&dev_distmod, memSizeF4*NC);
    hipMalloc((void**)&dev_delta_t, memSizeF4*NC);
    hipMalloc((void**)&dev_bonds, sizeof(uint2)*params.nb0);
    hipMalloc((void**)&dev_bonds_delta, sizeof(Float4)*params.nb0);
    hipMalloc((void**)&dev_bonds_omega, sizeof(Float4)*params.nb0);

    // Sorted arrays
    hipMalloc((void**)&dev_x_sorted, memSizeF4);
    hipMalloc((void**)&dev_vel_sorted, memSizeF4);
    hipMalloc((void**)&dev_angvel_sorted, memSizeF4);

    // Energy arrays
    hipMalloc((void**)&dev_es_dot, memSizeF);
    hipMalloc((void**)&dev_ev_dot, memSizeF);
    hipMalloc((void**)&dev_es, memSizeF);
    hipMalloc((void**)&dev_ev, memSizeF);
    hipMalloc((void**)&dev_p, memSizeF);

    // Cell-related arrays
    hipMalloc((void**)&dev_gridParticleCellID, sizeof(unsigned int)*np);
    hipMalloc((void**)&dev_gridParticleIndex, sizeof(unsigned int)*np);
    hipMalloc((void**)&dev_cellStart, sizeof(unsigned int)
               *grid.num[0]*grid.num[1]*grid.num[2]);
    hipMalloc((void**)&dev_cellEnd, sizeof(unsigned int)
               *grid.num[0]*grid.num[1]*grid.num[2]);

    // Host contact bookkeeping arrays
    k.contacts = new unsigned int[np*NC];
    // Initialize contacts lists to np
#pragma omp parallel for if(np>100)
    for (unsigned int i=0; i<(np*NC); ++i)
        k.contacts[i] = np;
    k.distmod = new Float4[np*NC];
    k.delta_t = new Float4[np*NC];

    // Wall arrays
    hipMalloc((void**)&dev_walls_wmode, sizeof(int)*walls.nw);
    hipMalloc((void**)&dev_walls_nx, sizeof(Float4)*walls.nw);
    hipMalloc((void**)&dev_walls_mvfd, sizeof(Float4)*walls.nw);
    hipMalloc((void**)&dev_walls_tau_x, sizeof(Float)*walls.nw);
    hipMalloc((void**)&dev_walls_tau_eff_x_pp, sizeof(Float)*walls.nw*np);
    hipMalloc((void**)&dev_walls_force_pp, sizeof(Float)*walls.nw*np);
    hipMalloc((void**)&dev_walls_acc, sizeof(Float)*walls.nw);
    // dev_walls_force_partial allocated later
    // dev_walls_tau_eff_x_partial allocated later

    checkForCudaErrors("End of allocateGlobalDeviceMemory");
    if (verbose == 1)
        std::cout << "Done" << std::endl;
}

// Allocate global memory on other devices required for "interact" function.
// The values of domain_size[ndevices] must be set beforehand.
void DEM::allocateHelperDeviceMemory(void)
{
    // Particle memory size
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // Initialize pointers to per-GPU arrays
    hdev_gridParticleIndex = (unsigned**)malloc(ndevices*sizeof(unsigned*));
    hdev_gridCellStart     = (unsigned**)malloc(ndevices*sizeof(unsigned*));
    hdev_gridCellEnd       = (unsigned**)malloc(ndevices*sizeof(unsigned*));
    hdev_x                 = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_x_sorted          = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_vel               = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_vel_sorted        = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_angvel            = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_angvel_sorted     = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_walls_nx          = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_walls_mvfd        = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_distmod           = (Float4**)malloc(ndevices*sizeof(Float4*));

    hdev_force             = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_torque            = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_delta_t           = (Float4**)malloc(ndevices*sizeof(Float4*));
    hdev_es_dot            = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_es                = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_ev_dot            = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_ev                = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_p                 = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_walls_force_pp    = (Float**)malloc(ndevices*sizeof(Float*));
    hdev_contacts          = (unsigned**)malloc(ndevices*sizeof(unsigned*));

    for (int d=0; d<ndevices; d++) {

        // do not allocate memory on primary GPU
        if (d == device)
            continue;

        hipSetDevice(d);

        // allocate space for full input arrays for interact()
        hipMalloc((void**)&hdev_gridParticleIndex[d], sizeof(unsigned int)*np);
        hipMalloc((void**)&hdev_gridCellStart[d], sizeof(unsigned int)
                   *grid.num[0]*grid.num[1]*grid.num[2]);
        hipMalloc((void**)&hdev_gridCellEnd[d], sizeof(unsigned int)
                   *grid.num[0]*grid.num[1]*grid.num[2]);
        hipMalloc((void**)&hdev_x[d], memSizeF4);
        hipMalloc((void**)&hdev_x_sorted[d], memSizeF4);
        hipMalloc((void**)&hdev_vel[d], memSizeF4);
        hipMalloc((void**)&hdev_vel_sorted[d], memSizeF4);
        hipMalloc((void**)&hdev_angvel[d], memSizeF4);
        hipMalloc((void**)&hdev_angvel_sorted[d], memSizeF4);
        hipMalloc((void**)&hdev_walls_nx[d], sizeof(Float4)*walls.nw);
        hipMalloc((void**)&hdev_walls_mvfd[d], sizeof(Float4)*walls.nw);
        hipMalloc((void**)&hdev_distmod[d], memSizeF4*NC);

        // allocate space for partial output arrays for interact()
        hipMalloc((void**)&hdev_force[d], sizeof(Float4)*domain_size[d]);
        hipMalloc((void**)&hdev_torque[d], sizeof(Float4)*domain_size[d]);
        hipMalloc((void**)&hdev_es_dot[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_ev_dot[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_es[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_ev[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_p[d], sizeof(Float)*domain_size[d]);
        hipMalloc((void**)&hdev_walls_force_pp[d],
                   sizeof(Float)*domain_size[d]*walls.nw);
        hipMalloc((void**)&hdev_contacts[d],
                   sizeof(unsigned)*domain_size[d]*NC);
        hipMalloc((void**)&hdev_delta_t[d], sizeof(Float4)*domain_size[d]*NC);

        checkForCudaErrors("During allocateGlobalDeviceMemoryOtherDevices");
    }
    hipSetDevice(device); // select main GPU
}

void DEM::freeHelperDeviceMemory()
{
    for (int d=0; d<ndevices; d++) {

        // do not allocate memory on primary GPU
        if (d == device)
            continue;

        hipSetDevice(d);

        hipFree(hdev_gridParticleIndex[d]);
        hipFree(hdev_gridCellStart[d]);
        hipFree(hdev_gridCellEnd[d]);
        hipFree(hdev_x[d]);
        hipFree(hdev_vel[d]);
        hipFree(hdev_vel_sorted[d]);
        hipFree(hdev_angvel[d]);
        hipFree(hdev_angvel_sorted[d]);
        hipFree(hdev_walls_nx[d]);
        hipFree(hdev_walls_mvfd[d]);
        hipFree(hdev_distmod[d]);

        hipFree(hdev_force[d]);
        hipFree(hdev_torque[d]);
        hipFree(hdev_es_dot[d]);
        hipFree(hdev_ev_dot[d]);
        hipFree(hdev_es[d]);
        hipFree(hdev_ev[d]);
        hipFree(hdev_p[d]);
        hipFree(hdev_walls_force_pp[d]);
        hipFree(hdev_contacts[d]);
        hipFree(hdev_delta_t[d]);

        checkForCudaErrors("During helper device hipFree calls");
    }
    hipSetDevice(device); // select primary GPU
}

void DEM::freeGlobalDeviceMemory()
{
    if (verbose == 1)
        printf("\nFreeing device memory:                           ");

    // Particle arrays
    hipFree(dev_x);
    hipFree(dev_xyzsum);
    hipFree(dev_vel);
    hipFree(dev_vel0);
    hipFree(dev_acc);
    hipFree(dev_force);
    hipFree(dev_angpos);
    hipFree(dev_angvel);
    hipFree(dev_angvel0);
    hipFree(dev_angacc);
    hipFree(dev_torque);

    hipFree(dev_contacts);
    hipFree(dev_distmod);
    hipFree(dev_delta_t);
    hipFree(dev_bonds);
    hipFree(dev_bonds_delta);
    hipFree(dev_bonds_omega);

    hipFree(dev_es_dot);
    hipFree(dev_es);
    hipFree(dev_ev_dot);
    hipFree(dev_ev);
    hipFree(dev_p);

    hipFree(dev_x_sorted);
    hipFree(dev_vel_sorted);
    hipFree(dev_angvel_sorted);

    // Cell-related arrays
    hipFree(dev_gridParticleIndex);
    hipFree(dev_cellStart);
    hipFree(dev_cellEnd);

    // Wall arrays
    hipFree(dev_walls_nx);
    hipFree(dev_walls_mvfd);
    hipFree(dev_walls_tau_x);
    hipFree(dev_walls_force_partial);
    hipFree(dev_walls_force_pp);
    hipFree(dev_walls_acc);
    hipFree(dev_walls_tau_eff_x_pp);
    hipFree(dev_walls_tau_eff_x_partial);

    // Fluid arrays
    if (fluid == 1 && cfd_solver == 0) {
        freeNSmemDev();
    }
    if (fluid == 1 && cfd_solver == 1) {
        freeDarcyMemDev();
    }

    //checkForCudaErrors("During hipFree calls");

    if (verbose == 1)
        std::cout << "Done" << std::endl;
}


void DEM::transferToGlobalDeviceMemory(int statusmsg)
{
    if (verbose == 1 && statusmsg == 1)
        std::cout << "  Transfering data to the device:                 ";

    // Commonly-used memory sizes
    unsigned int memSizeF  = sizeof(Float) * np;
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // Copy static-size structure data from host to global device memory
    //hipMemcpy(dev_time, &time, sizeof(Time), hipMemcpyHostToDevice);

    // Kinematic particle values
    hipMemcpy( dev_x,        k.x,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_xyzsum,   k.xyzsum,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_vel,      k.vel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_vel0,     k.vel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_acc,      k.acc,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_force,    k.force,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angpos,   k.angpos,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angvel,   k.angvel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angvel0,  k.angvel,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_angacc,   k.angacc,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_torque,   k.torque,
                memSizeF4, hipMemcpyHostToDevice);
    hipMemcpy( dev_contacts, k.contacts,
                sizeof(unsigned int)*np*NC, hipMemcpyHostToDevice);
    hipMemcpy( dev_distmod, k.distmod,
                memSizeF4*NC, hipMemcpyHostToDevice);
    hipMemcpy( dev_delta_t, k.delta_t,
                memSizeF4*NC, hipMemcpyHostToDevice);
    hipMemcpy( dev_bonds, k.bonds,
                sizeof(uint2)*params.nb0, hipMemcpyHostToDevice);
    hipMemcpy( dev_bonds_delta, k.bonds_delta,
                sizeof(Float4)*params.nb0, hipMemcpyHostToDevice);
    hipMemcpy( dev_bonds_omega, k.bonds_omega,
                sizeof(Float4)*params.nb0, hipMemcpyHostToDevice);

    // Individual particle energy values
    hipMemcpy( dev_es_dot, e.es_dot,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_es,     e.es,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_ev_dot, e.ev_dot,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_ev,     e.ev,
                memSizeF, hipMemcpyHostToDevice);
    hipMemcpy( dev_p, e.p,
                memSizeF, hipMemcpyHostToDevice);

    // Wall parameters
    hipMemcpy( dev_walls_wmode, walls.wmode,
                sizeof(int)*walls.nw, hipMemcpyHostToDevice);
    hipMemcpy( dev_walls_nx,    walls.nx,
                sizeof(Float4)*walls.nw, hipMemcpyHostToDevice);
    hipMemcpy( dev_walls_mvfd,  walls.mvfd,
                sizeof(Float4)*walls.nw, hipMemcpyHostToDevice);
    hipMemcpy( dev_walls_tau_x,  walls.tau_x,
                sizeof(Float)*walls.nw, hipMemcpyHostToDevice);

    // Fluid arrays
    if (fluid == 1) {
        if (cfd_solver == 0) {
            transferNStoGlobalDeviceMemory(1);
        } else if (cfd_solver == 1) {
            transferDarcyToGlobalDeviceMemory(1);
        } else {
            std::cerr << "Error: cfd_solver value not understood ("
                << cfd_solver << ")" << std::endl;
        }
    }

    checkForCudaErrors("End of transferToGlobalDeviceMemory");
    if (verbose == 1 && statusmsg == 1)
        std::cout << "Done" << std::endl;
}

void DEM::transferFromGlobalDeviceMemory()
{
    //std::cout << "  Transfering data from the device:               ";

    // Commonly-used memory sizes
    unsigned int memSizeF  = sizeof(Float) * np;
    unsigned int memSizeF4 = sizeof(Float4) * np;

    // Copy static-size structure data from host to global device memory
    //hipMemcpy(&time, dev_time, sizeof(Time), hipMemcpyDeviceToHost);

    // Kinematic particle values
    hipMemcpy( k.x, dev_x,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.xyzsum, dev_xyzsum,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.vel, dev_vel,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.acc, dev_acc,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.force, dev_force,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.angpos, dev_angpos,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.angvel, dev_angvel,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.angacc, dev_angacc,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.torque, dev_torque,
            memSizeF4, hipMemcpyDeviceToHost);
    hipMemcpy( k.contacts, dev_contacts,
            sizeof(unsigned int)*np*NC, hipMemcpyDeviceToHost);
    hipMemcpy( k.distmod, dev_distmod,
            memSizeF4*NC, hipMemcpyDeviceToHost);
    hipMemcpy( k.delta_t, dev_delta_t,
            memSizeF4*NC, hipMemcpyDeviceToHost);
    hipMemcpy( k.bonds, dev_bonds,
            sizeof(uint2)*params.nb0, hipMemcpyDeviceToHost);
    hipMemcpy( k.bonds_delta, dev_bonds_delta,
            sizeof(Float4)*params.nb0, hipMemcpyDeviceToHost);
    hipMemcpy( k.bonds_omega, dev_bonds_omega,
            sizeof(Float4)*params.nb0, hipMemcpyDeviceToHost);

    // Individual particle energy values
    hipMemcpy( e.es_dot, dev_es_dot,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.es, dev_es,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.ev_dot, dev_ev_dot,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.ev, dev_ev,
            memSizeF, hipMemcpyDeviceToHost);
    hipMemcpy( e.p, dev_p,
            memSizeF, hipMemcpyDeviceToHost);

    // Wall parameters
    hipMemcpy( walls.wmode, dev_walls_wmode,
            sizeof(int)*walls.nw, hipMemcpyDeviceToHost);
    hipMemcpy( walls.nx, dev_walls_nx,
            sizeof(Float4)*walls.nw, hipMemcpyDeviceToHost);
    hipMemcpy( walls.mvfd, dev_walls_mvfd,
            sizeof(Float4)*walls.nw, hipMemcpyDeviceToHost);
    hipMemcpy( walls.tau_x, dev_walls_tau_x,
            sizeof(Float)*walls.nw, hipMemcpyDeviceToHost);

    // Fluid arrays
    if (fluid == 1 && cfd_solver == 0) {
        transferNSfromGlobalDeviceMemory(0);
    }
    else if (fluid == 1 && cfd_solver == 1) {
        transferDarcyFromGlobalDeviceMemory(0);
        checkDarcyStability();
    }

    //checkForCudaErrors("End of transferFromGlobalDeviceMemory");
}


// Iterate through time by explicit time integration
void DEM::startTime()
{
    using std::cout;
    using std::cerr;
    using std::endl;

    std::string outfile;
    char file[200];
    FILE *fp;

    // Synchronization point
    hipDeviceSynchronize();
    checkForCudaErrors("Start of startTime()");

    // Write initial data to output/<sid>.output00000.bin
    if (time.step_count == 0)
        writebin(("output/" + sid + ".output00000.bin").c_str());

    // Time variables
    clock_t tic, toc;
    double filetimeclock, time_spent;
    float dev_time_spent;

    // Start CPU clock
    tic = clock();

    //// GPU workload configuration
    unsigned int threadsPerBlock = 256; 
    //unsigned int threadsPerBlock = 512; 

    // Create enough blocks to accomodate the particles
    unsigned int blocksPerGrid = iDivUp(np, threadsPerBlock); 
    dim3 dimGrid(blocksPerGrid, 1, 1); // Blocks arranged in 1D grid
    dim3 dimBlock(threadsPerBlock, 1, 1); // Threads arranged in 1D block

    unsigned int blocksPerGridBonds = iDivUp(params.nb0, threadsPerBlock); 
    dim3 dimGridBonds(blocksPerGridBonds, 1, 1); // Blocks arranged in 1D grid

    // Use 3D block and grid layout for cell-centered fluid calculations
    dim3 dimBlockFluid(8, 8, 8);    // 512 threads per block
    dim3 dimGridFluid(
            iDivUp(grid.num[0], dimBlockFluid.x),
            iDivUp(grid.num[1], dimBlockFluid.y),
            iDivUp(grid.num[2], dimBlockFluid.z));
    if (dimGridFluid.z > 64 && fluid == 1) {
        cerr << "Error: dimGridFluid.z > 64" << endl;
        exit(1);
    }

    // Use 3D block and grid layout for cell-face fluid calculations
    dim3 dimBlockFluidFace(8, 8, 8);    // 512 threads per block
    dim3 dimGridFluidFace(
            iDivUp(grid.num[0]+1, dimBlockFluidFace.x),
            iDivUp(grid.num[1]+1, dimBlockFluidFace.y),
            iDivUp(grid.num[2]+1, dimBlockFluidFace.z));
    if (dimGridFluidFace.z > 64 && fluid == 1) {
        cerr << "Error: dimGridFluidFace.z > 64" << endl;
        exit(1);
    }


    // Shared memory per block
    unsigned int smemSize = sizeof(unsigned int)*(threadsPerBlock+1);

    // Pre-sum of force per wall
    hipMalloc((void**)&dev_walls_force_partial,
            sizeof(Float)*dimGrid.x*walls.nw);

    // Pre-sum of shear stress per wall
    hipMalloc((void**)&dev_walls_tau_eff_x_partial,
            sizeof(Float)*dimGrid.x*walls.nw);

    // Report to stdout
    if (verbose == 1) {
        cout << "\n  Device memory allocation and transfer complete.\n"
            << "  - Blocks per grid: "
            << dimGrid.x << "*" << dimGrid.y << "*" << dimGrid.z << "\n"
            << "  - Threads per block: "
            << dimBlock.x << "*" << dimBlock.y << "*" << dimBlock.z << "\n"
            << "  - Shared memory required per block: " << smemSize << " bytes"
            << endl;
        if (fluid == 1) {
            cout << "  - Blocks per fluid grid: "
                << dimGridFluid.x << "*" << dimGridFluid.y << "*" <<
                dimGridFluid.z << "\n"
                << "  - Threads per fluid block: "
                << dimBlockFluid.x << "*" << dimBlockFluid.y << "*" <<
                dimBlockFluid.z << endl;
        }
    }

    // Initialize counter variable values
    filetimeclock = 0.0;
    long iter = 0;
    const int stdout_report = 10; // no of steps between reporting to stdout

    // Create first status.dat
    //sprintf(file,"output/%s.status.dat", sid);
    outfile = "output/" + sid + ".status.dat";
    fp = fopen(outfile.c_str(), "w");
    fprintf(fp,"%2.4e %2.4e %d\n", 
            time.current, 
            100.0*time.current/time.total, 
            time.step_count);
    fclose(fp);

    if (verbose == 1) {
        cout << "\n  Entering the main calculation time loop...\n\n"
            << "  IMPORTANT: Do not close this terminal, doing so will \n"
            << "             terminate this SPHERE process. Follow the \n"
            << "             progress by executing:\n"
            << "                $ ./sphere_status " << sid << endl << endl;
    }


    // Start GPU clock
    hipEvent_t dev_tic, dev_toc;
    hipEventCreate(&dev_tic);
    hipEventCreate(&dev_toc);
    hipEventRecord(dev_tic, 0);

    // If profiling is enabled, initialize timers for each kernel
    hipEvent_t kernel_tic, kernel_toc;
    float kernel_elapsed;
    double t_calcParticleCellID = 0.0;
    double t_thrustsort = 0.0;
    double t_reorderArrays = 0.0;
    double t_topology = 0.0;
    double t_interact = 0.0;
    double t_bondsLinear = 0.0;
    double t_latticeBoltzmannD3Q19 = 0.0;
    double t_integrate = 0.0;
    double t_summation = 0.0;
    double t_integrateWalls = 0.0;

    double t_findPorositiesDev = 0.0;
    double t_findNSstressTensor = 0.0;
    double t_findNSdivphiviv = 0.0;
    double t_findNSdivtau = 0.0;
    double t_findPredNSvelocities = 0.0;
    double t_setNSepsilon = 0.0;
    double t_setNSdirichlet = 0.0;
    double t_setNSghostNodesDev = 0.0;
    double t_findNSforcing = 0.0;
    double t_jacobiIterationNS = 0.0;
    double t_updateNSvelocityPressure = 0.0;

    double t_findDarcyPorosities = 0.0;
    double t_setDarcyGhostNodes = 0.0;
    double t_findDarcyPressureForce = 0.0;
    double t_setDarcyTopPressure = 0.0;
    double t_findDarcyPermeabilities = 0.0;
    double t_findDarcyPermeabilityGradients = 0.0;
    //double t_findDarcyPressureChange = 0.0;
    double t_updateDarcySolution = 0.0;
    double t_copyValues = 0.0;
    double t_findDarcyVelocities = 0.0;

    if (PROFILING == 1) {
        hipEventCreate(&kernel_tic);
        hipEventCreate(&kernel_toc);
    }

    // The model start time is saved for profiling performance
    double t_start = time.current;
    double t_ratio;     // ration between time flow in model vs. reality

    // Hard-coded parameters for stepwise velocity change (rate-state exp)
    int velocity_state = 1;  // 1: v1, 2: v2
    int change_velocity_state = 0;  // 1: increase velocity, 2: decrease vel.
    const Float velocity_factor = 10.0;  // v2 = v1*velocity_factor
    const Float v2_start = 10.0; // seconds
    const Float v2_end = 15.0;  // seconds

    // Index of dynamic top wall (if it exists)
    unsigned int wall0_iz = 10000000;
    // weight of fluid between two cells in z direction
    Float dp_dz;
    if (fluid == 1) {
        if (cfd_solver == 0)
            dp_dz = fabs(ns.rho_f*params.g[2]*grid.L[2]/grid.num[2]);
        else if (cfd_solver == 1) {
            dp_dz = fabs(darcy.rho_f*params.g[2]*grid.L[2]/grid.num[2]);

            // determine pressure at top wall at t=0
            darcy.p_top_orig = darcy.p[d_idx(0,0,darcy.nz-1)]
                                - darcy.p_mod_A
                                *sin(2.0*M_PI*darcy.p_mod_f*time.current
                                        + darcy.p_mod_phi);
        }
    }
    //std::cout << "dp_dz = " << dp_dz << std::endl;

    // Write a log file of the number of iterations it took before
    // convergence in the fluid solver
    std::ofstream convlog;
    if (write_conv_log == 1) {
        std::string f = "output/" + sid + "-conv.log";
        convlog.open(f.c_str());
    }

    if (verbose == 1)
        cout << "  Current simulation time: " << time.current << " s.";

    // MAIN CALCULATION TIME LOOP
    while (time.current <= time.total) {

        // Print current step number to terminal
        //printf("\n\n@@@ DEM time step: %ld\n", iter);

        // Routine check for errors
        checkForCudaErrors("Start of main while loop");

        if (np > 0) {

            // check if particle positions have finite values
#ifdef CHECK_PARTICLES_FINITE
            checkParticlePositions<<<dimGrid, dimBlock>>>(dev_x);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post checkParticlePositions", iter);
#endif

            // If the grid is adaptive, readjust the grid height to equal the 
            // positions of the dynamic walls
            if (grid.adaptive == 1 && walls.nw > 0) {
                updateGridSize();
            }

            // For each particle: 
            // Compute hash key (cell index) from position 
            // in the fine, uniform and homogenous grid.
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            calcParticleCellID<<<dimGrid, dimBlock>>>(dev_gridParticleCellID,
                    dev_gridParticleIndex, 
                    dev_x);

            // Synchronization point
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_calcParticleCellID);
            checkForCudaErrorsIter("Post calcParticleCellID", iter);


            // Sort particle (key, particle ID) pairs by hash key with Thrust
            // radix sort
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            thrust::sort_by_key(
                    thrust::device_ptr<uint>(dev_gridParticleCellID),
                    thrust::device_ptr<uint>(dev_gridParticleCellID + np),
                    thrust::device_ptr<uint>(dev_gridParticleIndex));
            hipDeviceSynchronize(); // Maybe Thrust synchronizes implicitly?
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_thrustsort);
            checkForCudaErrorsIter("Post thrust::sort_by_key", iter);


            // Zero cell array values by setting cellStart to its highest
            // possible value, specified with pointer value 0xffffffff, which
            // for a 32 bit unsigned int is 4294967295.
            hipMemset(dev_cellStart, 0xffffffff, 
                    grid.num[0]*grid.num[1]*grid.num[2]*sizeof(unsigned int));
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post hipMemset", iter);

            // Use sorted order to reorder particle arrays (position,
            // velocities, radii) to ensure coherent memory access. Save ordered
            // configurations in new arrays (*_sorted).
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            reorderArrays<<<dimGrid, dimBlock, smemSize>>>(dev_cellStart, 
                    dev_cellEnd,
                    dev_gridParticleCellID, 
                    dev_gridParticleIndex,
                    dev_x, dev_vel, 
                    dev_angvel,
                    dev_x_sorted, 
                    dev_vel_sorted, 
                    dev_angvel_sorted);

            // Synchronization point
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_reorderArrays);
            checkForCudaErrorsIter("Post reorderArrays", iter);

            // The contact search in topology() is only necessary for
            // determining the accumulated shear distance needed in the linear
            // elastic and nonlinear contact force model
            if (params.contactmodel == 2 || params.contactmodel == 3) {
                // For each particle: Search contacts in neighbor cells
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                topology<<<dimGrid, dimBlock>>>(dev_cellStart, 
                        dev_cellEnd,
                        dev_gridParticleIndex,
                        dev_x_sorted, 
                        dev_contacts,
                        dev_distmod);

                // Synchronization point
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_topology);
                checkForCudaErrorsIter(
                        "Post topology: One or more particles moved "
                        "outside the grid.\nThis could possibly be caused by a "
                        "numerical instability.\nIs the computational time step"
                        " too large?", iter);
            }

            // For each particle process collisions and compute resulting forces
            //cudaPrintfInit();
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            interact<<<dimGrid, dimBlock>>>(dev_gridParticleIndex,
                    dev_cellStart,
                    dev_cellEnd,
                    dev_x,
                    dev_x_sorted,
                    dev_vel_sorted,
                    dev_angvel_sorted,
                    dev_vel,
                    dev_angvel,
                    dev_force, 
                    dev_torque, 
                    dev_es_dot,
                    dev_ev_dot, 
                    dev_es,
                    dev_ev,
                    dev_p,
                    dev_walls_nx,
                    dev_walls_mvfd,
                    dev_walls_force_pp,
                    dev_contacts,
                    dev_distmod,
                    dev_delta_t);

            // Synchronization point
            hipDeviceSynchronize();
            //cudaPrintfDisplay(stdout, true);
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_interact);
            checkForCudaErrorsIter(
                    "Post interact - often caused if particles move "
                    "outside the grid", iter);

            // Process particle pairs
            if (params.nb0 > 0) {
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                bondsLinear<<<dimGridBonds, dimBlock>>>(
                        dev_bonds,
                        dev_bonds_delta,
                        dev_bonds_omega,
                        dev_x,
                        dev_vel,
                        dev_angvel,
                        dev_force,
                        dev_torque);
                // Synchronization point
                hipDeviceSynchronize();
                //cudaPrintfDisplay(stdout, true);
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_bondsLinear);
                checkForCudaErrorsIter("Post bondsLinear", iter);
            }
        }

        // Solve fluid flow through the grid
        if (fluid == 1) {

            // Navier-Stokes solution
            if (cfd_solver == 0) {

                checkForCudaErrorsIter("Before findPorositiesDev", iter);
                // Find cell porosities, average particle velocities, and
                // average particle diameters. These are needed for predicting
                // the fluid velocities
                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                findPorositiesVelocitiesDiametersSpherical
                //findPorositiesVelocitiesDiametersSphericalGradient
                    <<<dimGridFluid, dimBlockFluid>>>(
                            dev_cellStart,
                            dev_cellEnd,
                            dev_x_sorted,
                            dev_vel_sorted,
                            dev_ns_phi,
                            dev_ns_dphi,
                            dev_ns_vp_avg,
                            dev_ns_d_avg,
                            iter,
                            np,
                            ns.c_phi);
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_findPorositiesDev);
                checkForCudaErrorsIter("Post findPorositiesDev", iter);

#ifdef CFD_DEM_COUPLING
                /*if (params.nu <= 0.0) {
                  std::cerr << "Error! The fluid needs a positive viscosity "
                  "value in order to simulate particle-fluid interaction."
                  << std::endl;
                  exit(1);
                  }*/
                if (iter == 0) {
                    // set cell center ghost nodes
                    setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_v, ns.bc_bot, ns.bc_top);

                    // find cell face velocities
                    interpolateCenterToFace
                        <<<dimGridFluidFace, dimBlockFluidFace>>>(
                                dev_ns_v,
                                dev_ns_v_x,
                                dev_ns_v_y,
                                dev_ns_v_z);
                    hipDeviceSynchronize();
                    checkForCudaErrors("Post interpolateCenterToFace");
                }

                setNSghostNodesFace<Float>
                    <<<dimGridFluidFace, dimBlockFluidFace>>>(
                            dev_ns_v_x,
                            dev_ns_v_y,
                            dev_ns_v_z,
                            ns.bc_bot,
                            ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodesFace", iter);

                findFaceDivTau<<<dimGridFluidFace, dimBlockFluidFace>>>(
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z,
                        ns.mu,
                        dev_ns_div_tau_x,
                        dev_ns_div_tau_y,
                        dev_ns_div_tau_z);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post findFaceDivTau", iter);

                setNSghostNodesFace<Float>
                    <<<dimGridFluidFace, dimBlockFluid>>>(
                            dev_ns_div_tau_x,
                            dev_ns_div_tau_y,
                            dev_ns_div_tau_z,
                            ns.bc_bot,
                            ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_div_tau)",
                        iter);

                setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_p, ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_p)", iter);

                setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_phi, ns.bc_bot, ns.bc_top);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_p)", iter);


                if (np > 0) {

                    // Per particle, find the fluid-particle interaction force
                    // f_pf and apply it to the particle
                    findInteractionForce<<<dimGrid, dimBlock>>>(
                            dev_x,
                            dev_vel,
                            dev_ns_phi,
                            dev_ns_p,
                            dev_ns_v_x,
                            dev_ns_v_y,
                            dev_ns_v_z,
                            dev_ns_div_tau_x,
                            dev_ns_div_tau_y,
                            dev_ns_div_tau_z,
                            //ns.c_v,
                            ns.mu,
                            ns.rho_f,
                            dev_ns_f_pf,
                            dev_force,
                            dev_ns_f_d,
                            dev_ns_f_p,
                            dev_ns_f_v,
                            dev_ns_f_sum);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post findInteractionForce", iter);

                    setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_p, ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSghostNodes(dev_ns_p)",
                            iter);

                    // Apply fluid-particle interaction force to the fluid
                    applyInteractionForceToFluid
                        <<<dimGridFluid, dimBlockFluid>>>(
                            dev_gridParticleIndex,
                            dev_cellStart,
                            dev_cellEnd,
                            dev_ns_f_pf,
                            dev_ns_F_pf);
                    //dev_ns_F_pf_x,
                    //dev_ns_F_pf_y,
                    //dev_ns_F_pf_z);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post applyInteractionForceToFluid",
                            iter);

                    setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_F_pf, ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSghostNodes(F_pf)", iter);
                }
#endif

                if ((iter % ns.ndem) == 0) {
                    // Initial guess for the top epsilon values. These may be
                    // changed in setUpperPressureNS
                    // TODO: Check if this should only be set when top bc=Dirichlet
                    Float pressure = ns.p[idx(0,0,ns.nz-1)];
                    Float pressure_new = pressure; // Dirichlet
                    Float epsilon_value = pressure_new - ns.beta*pressure;
                    setNSepsilonTop<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            dev_ns_epsilon_new,
                            epsilon_value);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSepsilonTop", iter);

#if defined(REPORT_EPSILON) || defined(REPORT_V_P_COMPONENTS) || defined(REPORT_V_C_COMPONENTS)
                    std::cout
                        << "\n\n@@@@@@ TIME STEP " << iter << " @@@"
                        << std::endl;
#endif

                    // find cell containing top wall
                    if (walls.nw > 0 &&
                            (walls.wmode[0] == 1 || walls.wmode[0] == 3)) {
                        wall0_iz = walls.nx->w/(grid.L[2]/grid.num[2]);
                        setNSepsilonAtTopWall<<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon,
                                dev_ns_epsilon_new,
                                wall0_iz,
                                epsilon_value,
                                dp_dz);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setNSepsilonAtTopWall",
                                iter);

#ifdef REPORT_EPSILON
                        std::cout
                            << "\n###### EPSILON setNSepsilonAtTopWall "
                            << "######" << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif
                    }

                    // Modulate the pressures at the upper boundary cells
                    if ((ns.p_mod_A > 1.0e-5 || ns.p_mod_A < -1.0e-5) &&
                            ns.p_mod_f > 1.0e-7) {
                        // original pressure
                        Float new_pressure = ns.p[idx(0,0,ns.nz-1)]
                            + ns.p_mod_A*sin(2.0*M_PI*ns.p_mod_f*time.current
                                    + ns.p_mod_phi);
                        setUpperPressureNS<<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_p,
                                dev_ns_epsilon,
                                dev_ns_epsilon_new,
                                ns.beta,
                                new_pressure);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setUpperPressureNS", iter);

#ifdef REPORT_MORE_EPSILON
                        std::cout
                            << "\n@@@@@@ TIME STEP " << iter << " @@@@@@"
                            << "\n###### EPSILON AFTER setUpperPressureNS "
                            << "######" << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif
                    }

                    // Set the values of the ghost nodes in the grid
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);

                    setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_p, ns.bc_bot, ns.bc_top);

                    //setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                    //dev_ns_v, ns.bc_bot, ns.bc_top);

                    setNSghostNodesFace<Float>
                        <<<dimGridFluidFace, dimBlockFluidFace>>>(
                                dev_ns_v_p_x,
                                dev_ns_v_p_y,
                                dev_ns_v_p_z,
                                ns.bc_bot, ns.bc_top);

                    setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_phi, ns.bc_bot, ns.bc_top);

                    setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_dphi, ns.bc_bot, ns.bc_top);

                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setNSghostNodesDev);
                    checkForCudaErrorsIter("Post setNSghostNodesDev", iter);
                    /*std::cout
                      << "\n###### EPSILON AFTER setNSghostNodesDev #####"
                      << std::endl;
                      transferNSepsilonFromGlobalDeviceMemory();
                      printNSarray(stdout, ns.epsilon, "epsilon");*/

                    // interpolate velocities to cell centers which makes
                    // velocity prediction easier
                    interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_v_x,
                            dev_ns_v_y,
                            dev_ns_v_z,
                            dev_ns_v);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter(
                            "Post interpolateFaceToCenter", iter);

                    // Set cell-center velocity ghost nodes
                    setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_v, ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSghostNodes(v)", iter);

                    // Find the divergence of phi*vi*v, needed for predicting
                    // the fluid velocities
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findNSdivphiviv<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_phi,
                            dev_ns_v,
                            dev_ns_div_phi_vi_v);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findNSdivphiviv);
                    checkForCudaErrorsIter("Post findNSdivphiviv", iter);

                    // Set cell-center ghost nodes
                    setNSghostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_div_phi_vi_v, ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post setNSghostNodes(div_phi_vi_v)",
                            iter);

                    // Predict the fluid velocities on the base of the old
                    // pressure field and ignoring the incompressibility
                    // constraint
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findPredNSvelocities<<<dimGridFluidFace, dimBlockFluidFace>>>(
                            dev_ns_p,
                            dev_ns_v_x,
                            dev_ns_v_y,
                            dev_ns_v_z,
                            dev_ns_phi,
                            dev_ns_dphi,
                            dev_ns_div_tau_x,
                            dev_ns_div_tau_y,
                            dev_ns_div_tau_z,
                            dev_ns_div_phi_vi_v,
                            ns.bc_bot,
                            ns.bc_top,
                            ns.beta,
                            dev_ns_F_pf,
                            ns.ndem,
                            wall0_iz,
                            ns.c_v,
                            ns.mu,
                            ns.rho_f,
                            dev_ns_v_p_x,
                            dev_ns_v_p_y,
                            dev_ns_v_p_z);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findPredNSvelocities);
                    checkForCudaErrorsIter("Post findPredNSvelocities", iter);

                    setNSghostNodesFace<Float>
                        <<<dimGridFluidFace, dimBlockFluidFace>>>(
                                dev_ns_v_p_x,
                                dev_ns_v_p_y,
                                dev_ns_v_p_z,
                                ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter(
                            "Post setNSghostNodesFace(dev_ns_v_p)", iter);

                    interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_v_p_x,
                            dev_ns_v_p_y,
                            dev_ns_v_p_z,
                            dev_ns_v_p);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter(
                            "Post interpolateFaceToCenter", iter);


                    // In the first iteration of the sphere program, we'll need
                    // to manually estimate the values of epsilon. In the
                    // subsequent iterations, the previous values are  used.
                    if (iter == 0) {

                        // Define the first estimate of the values of epsilon.
                        // The initial guess depends on the value of ns.beta.
                        Float pressure = ns.p[idx(2,2,2)];
                        Float pressure_new = pressure; // Guess p_curr = p_new
                        Float epsilon_value = pressure_new - ns.beta*pressure;
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        setNSepsilonInterior<<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon, epsilon_value);
                        hipDeviceSynchronize();

                        setNSnormZero<<<dimGridFluid, dimBlockFluid>>>
                            (dev_ns_norm);
                        hipDeviceSynchronize();

                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_setNSepsilon);
                        checkForCudaErrorsIter("Post setNSepsilonInterior",
                                iter);

#ifdef REPORT_MORE_EPSILON
                        std::cout
                            << "\n###### EPSILON AFTER setNSepsilonInterior "
                            << "######" << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                        // Set the epsilon values at the lower boundary
                        pressure = ns.p[idx(0,0,0)];
                        pressure_new = pressure; // Guess p_current = p_new
                        epsilon_value = pressure_new - ns.beta*pressure;
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        setNSepsilonBottom<<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon,
                                dev_ns_epsilon_new,
                                epsilon_value);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_setNSdirichlet);
                        checkForCudaErrorsIter("Post setNSepsilonBottom", iter);

#ifdef REPORT_MORE_EPSILON
                        std::cout
                            << "\n###### EPSILON AFTER setNSepsilonBottom "
                            << "######" << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                        /*setNSghostNodes<Float>
                          <<<dimGridFluid, dimBlockFluid>>>(
                          dev_ns_epsilon);
                          hipDeviceSynchronize();
                          checkForCudaErrors(
                          "Post setNSghostNodesFloat(dev_ns_epsilon)",
                          iter);*/
                        setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon,
                                ns.bc_bot, ns.bc_top);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setNSghostNodesEpsilon(1)",
                                iter);

#ifdef REPORT_MORE_EPSILON
                        std::cout <<
                            "\n###### EPSILON AFTER setNSghostNodes(epsilon) "
                            << "######" << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif
                    }

                    // Solve the system of epsilon using a Jacobi iterative
                    // solver.  The average normalized residual is initialized
                    // to a large value.
                    //double avg_norm_res;
                    double max_norm_res;

                    // Write a log file of the normalized residuals during the
                    // Jacobi iterations
                    std::ofstream reslog;
                    if (write_res_log == 1)
                        reslog.open("max_res_norm.dat");

                    // transfer normalized residuals from GPU to CPU
#ifdef REPORT_MORE_EPSILON
                    std::cout << "\n###### BEFORE FIRST JACOBI ITERATION ######"
                        << "\n@@@@@@ TIME STEP " << iter << " @@@@@@"
                        << std::endl;
                    transferNSepsilonFromGlobalDeviceMemory();
                    printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                    for (unsigned int nijac = 0; nijac<ns.maxiter; ++nijac) {

                        // Only grad(epsilon) changes during the Jacobi
                        // iterations.  The remaining terms of the forcing
                        // function are only calculated during the first
                        // iteration.
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        findNSforcing<<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon,
                                dev_ns_phi,
                                dev_ns_dphi,
                                dev_ns_v_p,
                                dev_ns_v_p_x,
                                dev_ns_v_p_y,
                                dev_ns_v_p_z,
                                nijac,
                                ns.ndem,
                                ns.c_v,
                                ns.rho_f,
                                dev_ns_f1,
                                dev_ns_f2,
                                dev_ns_f);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_findNSforcing);
                        checkForCudaErrorsIter("Post findNSforcing", iter);
                        /*setNSghostNodesForcing
                          <<dimGridFluid, dimBlockFluid>>>(
                          dev_ns_f1,
                          dev_ns_f2,
                          dev_ns_f,
                          nijac);
                          hipDeviceSynchronize();
                          checkForCudaErrors("Post setNSghostNodesForcing",
                          iter);*/

                        setNSghostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon,
                                ns.bc_bot, ns.bc_top);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setNSghostNodesEpsilon(2)",
                                iter);

#ifdef REPORT_EPSILON
                        std::cout << "\n###### JACOBI ITERATION "
                            << nijac
                            << " after setNSghostNodes(epsilon,2) ######"
                            << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                        // Perform a single Jacobi iteration
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        jacobiIterationNS<<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon,
                                dev_ns_epsilon_new,
                                dev_ns_norm,
                                dev_ns_f,
                                ns.bc_bot,
                                ns.bc_top,
                                ns.theta,
                                wall0_iz,
                                dp_dz);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_jacobiIterationNS);
                        checkForCudaErrorsIter("Post jacobiIterationNS", iter);

                        // set Dirichlet and Neumann BC at cells containing top
                        // wall
                        /*if (walls.nw > 0 && walls.wmode[0] == 1) {
                          setNSepsilonAtTopWall
                          <<<dimGridFluid, dimBlockFluid>>>(
                          dev_ns_epsilon,
                          dev_ns_epsilon_new,
                          wall0_iz,
                          epsilon_value,
                          dp_dz);
                          hipDeviceSynchronize();
                          checkForCudaErrorsIter("Post setNSepsilonAtTopWall",
                          iter);
                          }*/

                        // Copy new values to current values
                        copyValues<Float><<<dimGridFluid, dimBlockFluid>>>(
                                dev_ns_epsilon_new,
                                dev_ns_epsilon);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter
                            ("Post copyValues (epsilon_new->epsilon)", iter);

#ifdef REPORT_EPSILON
                        std::cout << "\n###### JACOBI ITERATION "
                            << nijac << " after jacobiIterationNS ######"
                            << std::endl;
                        transferNSepsilonFromGlobalDeviceMemory();
                        printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                        if (nijac % nijacnorm == 0) {

                            // Read the normalized residuals from the device
                            transferNSnormFromGlobalDeviceMemory();

                            // Write the normalized residuals to the terminal
                            //printNSarray(stdout, ns.norm, "norm");

                            // Find the maximum value of the normalized
                            // residuals
                            max_norm_res = maxNormResNS();

                            // Write the Jacobi iteration number and maximum
                            // value of the normalized residual to the log file
                            if (write_res_log == 1)
                                reslog << nijac << '\t' << max_norm_res
                                    << std::endl;
                        }

                        if (max_norm_res < ns.tolerance) {

                            if (write_conv_log == 1
                                    && iter % conv_log_interval == 0)
                                convlog << iter+1 << '\t' << nijac << std::endl;

                            setNSghostNodes<Float>
                                <<<dimGridFluid, dimBlockFluid>>>(
                                        dev_ns_epsilon,
                                        ns.bc_bot, ns.bc_top);
                            hipDeviceSynchronize();
                            checkForCudaErrorsIter
                                ("Post setNSghostNodesEpsilon(4)", iter);

                            // Apply smoothing if requested
                            if (ns.gamma > 0.0) {

                                smoothing<<<dimGridFluid, dimBlockFluid>>>(
                                        dev_ns_epsilon,
                                        ns.gamma,
                                        ns.bc_bot, ns.bc_top);
                                hipDeviceSynchronize();
                                checkForCudaErrorsIter("Post smoothing", iter);

                                setNSghostNodes<Float>
                                    <<<dimGridFluid, dimBlockFluid>>>(
                                            dev_ns_epsilon,
                                            ns.bc_bot, ns.bc_top);
                                hipDeviceSynchronize();
                                checkForCudaErrorsIter
                                    ("Post setNSghostNodesEpsilon(4)", iter);
                            }

#ifdef REPORT_EPSILON
                            std::cout << "\n###### JACOBI ITERATION "
                                << nijac << " after smoothing ######"
                                << std::endl;
                            transferNSepsilonFromGlobalDeviceMemory();
                            printNSarray(stdout, ns.epsilon, "epsilon");
#endif

                            break;  // solution has converged, exit Jacobi loop
                        }

                        if (nijac >= ns.maxiter-1) {

                            if (write_conv_log == 1)
                                convlog << iter+1 << '\t' << nijac << std::endl;

                            std::cerr << "\nIteration " << iter << ", time " 
                                << iter*time.dt << " s: "
                                "Error, the epsilon solution in the fluid "
                                "calculations did not converge. Try increasing "
                                "the value of 'ns.maxiter' (" << ns.maxiter
                                << ") or increase 'ns.tolerance' ("
                                << ns.tolerance << ")." << std::endl;
                        }
                        //break; // end after Jacobi first iteration
                    } // end Jacobi iteration loop

                    if (write_res_log == 1)
                        reslog.close();

                    // Find the new pressures and velocities
                    if (PROFILING == 1)
                        startTimer(&kernel_tic);

                    updateNSpressure<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_epsilon,
                            ns.beta,
                            dev_ns_p);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post updateNSpressure", iter);

                    updateNSvelocity<<<dimGridFluidFace, dimBlockFluidFace>>>(
                            dev_ns_v_p_x,
                            dev_ns_v_p_y,
                            dev_ns_v_p_z,
                            dev_ns_phi,
                            dev_ns_epsilon,
                            ns.beta,
                            ns.bc_bot,
                            ns.bc_top,
                            ns.ndem,
                            ns.c_v,
                            ns.rho_f,
                            wall0_iz,
                            iter,
                            dev_ns_v_x,
                            dev_ns_v_y,
                            dev_ns_v_z);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_updateNSvelocityPressure);
                    checkForCudaErrorsIter("Post updateNSvelocity", iter);

                    setNSghostNodesFace<Float>
                        <<<dimGridFluidFace, dimBlockFluidFace>>>(
                                dev_ns_v_p_x,
                                dev_ns_v_p_y,
                                dev_ns_v_p_z,
                                ns.bc_bot, ns.bc_top);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter(
                            "Post setNSghostNodesFace(dev_ns_v)", iter);

                    interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                            dev_ns_v_x,
                            dev_ns_v_y,
                            dev_ns_v_z,
                            dev_ns_v);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("Post interpolateFaceToCenter",
                            iter);
                } // end iter % ns.dem == 0
            } // end cfd_solver == 0

            // Darcy solution
            else if (cfd_solver == 1) { 

#if defined(REPORT_EPSILON) || defined(REPORT_FORCING_TERMS)
                std::cout << "\n\n@@@@@@ TIME STEP " << iter << " @@@"
                        << std::endl;
#endif

                if (walls.nw > 0 &&
                        (walls.wmode[0] == 1 || walls.wmode[0] == 3)) {
                    wall0_iz = walls.nx->w/(grid.L[2]/grid.num[2]);
                }

                if (np > 0) {

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    setDarcyGhostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_darcy_p,
                            darcy.bc_xn, darcy.bc_xp,
                            darcy.bc_yn, darcy.bc_yp,
                            darcy.bc_bot, darcy.bc_top);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setDarcyGhostNodes);
                    checkForCudaErrorsIter("Post setDarcyGhostNodes("
                            "dev_darcy_p) before findDarcyPressureForce", iter);

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findDarcyPressureGradient<<<dimGridFluid, dimBlockFluid>>>(
                            dev_darcy_p,
                            dev_darcy_grad_p);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter("After findDarcyPressureGradient",
                            iter);

                    setDarcyGhostNodes<Float3><<<dimGridFluid, dimBlockFluid>>>(
                            dev_darcy_grad_p,
                            darcy.bc_xn, darcy.bc_xp,
                            darcy.bc_yn, darcy.bc_yp,
                            darcy.bc_bot, darcy.bc_top);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setDarcyGhostNodes);
                    checkForCudaErrorsIter("Post setDarcyGhostNodes("
                            "dev_darcy_grad_p)", iter);

                    /*if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findDarcyPorositiesLinear<<<dimGridFluid, dimBlockFluid>>>(
                            dev_cellStart,
                            dev_cellEnd,
                            dev_x_sorted,
                            dev_vel_sorted,
                            iter,
                            darcy.ndem,
                            np,
                            darcy.c_phi,
                            dev_darcy_phi,
                            dev_darcy_dphi,
                            dev_darcy_div_v_p);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findDarcyPorosities);
                    checkForCudaErrorsIter("Post findDarcyPorosities", iter);*/

                    /*findDarcyPressureForce<<<dimGrid, dimBlock>>>(
                            dev_x,
                            dev_darcy_p,
                            wall0_iz,
                            darcy.rho_f,
                            dev_force,
                            dev_darcy_f_p);*/
                    findDarcyPressureForceLinear<<<dimGrid, dimBlock>>>(
                            dev_x,
                            dev_darcy_grad_p,
                            dev_darcy_phi,
                            wall0_iz,
                            darcy.rho_f,
                            darcy.bc_top,
                            dev_force,
                            dev_darcy_f_p);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findDarcyPressureForce);
                    checkForCudaErrorsIter("Post findDarcyPressureForce",
                            iter);
                }

                if ((iter % darcy.ndem) == 0) {

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    /*findDarcyPorosities<<<dimGridFluid, dimBlockFluid>>>(
                            dev_cellStart,
                            dev_cellEnd,
                            dev_x_sorted,
                            dev_vel_sorted,
                            iter,
                            darcy.ndem,
                            np,
                            darcy.c_phi,
                            dev_darcy_phi,
                            dev_darcy_dphi);*/
                    findDarcyPorositiesLinear<<<dimGridFluid, dimBlockFluid>>>(
                            dev_cellStart,
                            dev_cellEnd,
                            dev_x_sorted,
                            dev_vel_sorted,
                            iter,
                            darcy.ndem,
                            np,
                            darcy.c_phi,
                            dev_darcy_phi,
                            dev_darcy_dphi,
                            dev_darcy_div_v_p,
                            dev_darcy_vp_avg);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findDarcyPorosities);
                    checkForCudaErrorsIter("Post findDarcyPorosities", iter);

                    // copy porosities to the frictionless Y boundaries
                    if (grid.periodic == 2) {
                        copyDarcyPorositiesToEdges<<<dimGridFluid, 
                            dimBlockFluid>>>(
                                dev_darcy_phi,
                                dev_darcy_dphi,
                                dev_darcy_div_v_p,
                                dev_darcy_vp_avg);
                        hipDeviceSynchronize();
                    }

                    // copy porosities to the frictionless lower Z boundary
                    if (grid.periodic == 2) {
                        copyDarcyPorositiesToBottom<<<dimGridFluid, 
                                dimBlockFluid>>>(
                                dev_darcy_phi,
                                dev_darcy_dphi,
                                dev_darcy_div_v_p,
                                dev_darcy_vp_avg);
                        hipDeviceSynchronize();
                    }

                    // Modulate the pressures at the upper boundary cells
                    if ((darcy.p_mod_A > 1.0e-5 || darcy.p_mod_A < -1.0e-5) &&
                            darcy.p_mod_f > 1.0e-7) {
                        // original pressure
                        Float new_pressure =
                            darcy.p_top_orig + darcy.p_mod_A
                            *sin(2.0*M_PI*darcy.p_mod_f*time.current
                                    + darcy.p_mod_phi);
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        setDarcyTopPressure<<<dimGridFluid, dimBlockFluid>>>(
                                new_pressure,
                                dev_darcy_p,
                                wall0_iz);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setUpperPressureNS", iter);

                        // Modulate the pressures at the top wall
                        setDarcyTopWallPressure
                            <<<dimGridFluid, dimBlockFluid>>>(
                                    new_pressure,
                                    wall0_iz,
                                    dev_darcy_p);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setDarcyTopWallPressure",
                                iter);

                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_setDarcyTopPressure);
                    }

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findDarcyPermeabilities<<<dimGridFluid, dimBlockFluid>>>(
                            darcy.k_c, dev_darcy_phi, dev_darcy_k);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findDarcyPermeabilities);
                    checkForCudaErrorsIter("Post findDarcyPermeabilities",
                            iter);

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    setDarcyGhostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_darcy_phi,
                            darcy.bc_xn, darcy.bc_xp,
                            darcy.bc_yn, darcy.bc_yp,
                            darcy.bc_bot, darcy.bc_top);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setDarcyGhostNodes);
                    checkForCudaErrorsIter(
                            "Post setDarcyGhostNodes(dev_darcy_phi)", iter);

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    setDarcyGhostNodes<Float><<<dimGridFluid, dimBlockFluid>>>(
                            dev_darcy_k,
                            darcy.bc_xn, darcy.bc_xp,
                            darcy.bc_yn, darcy.bc_yp,
                            darcy.bc_bot, darcy.bc_top);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setDarcyGhostNodes);
                    checkForCudaErrorsIter(
                            "Post setDarcyGhostNodes(dev_darcy_k)", iter);

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findDarcyPermeabilityGradients
                        <<<dimGridFluid, dimBlockFluid>>>
                        (dev_darcy_k, dev_darcy_grad_k);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findDarcyPermeabilityGradients);
                    checkForCudaErrorsIter(
                            "Post findDarcyPermeabilityGradients", iter);

                    if (iter == 0) {
                        setDarcyNormZero<<<dimGridFluid, dimBlockFluid>>>(
                                dev_darcy_norm);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter("Post setDarcyNormZero", iter);

                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        copyValues<Float><<<dimGridFluid, dimBlockFluid>>>(
                                dev_darcy_p,
                                dev_darcy_p_old);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_copyValues);
                        checkForCudaErrorsIter("Post copyValues(p -> p_old)",
                                iter);
                    }

                    if (darcy.bc_bot == 4 || darcy.bc_top == 4) {
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        setDarcyGhostNodesFlux<Float>
                            <<<dimGridFluid, dimBlockFluid>>>(
                                dev_darcy_p,
                                darcy.bc_bot,
                                darcy.bc_top,
                                darcy.bc_bot_flux,
                                darcy.bc_top_flux,
                                dev_darcy_k,
                                darcy.mu);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_setDarcyGhostNodes);
                        checkForCudaErrorsIter(
                                "Post setDarcyGhostNodesFlux", iter);
                    }

                    // Solve the system of epsilon using a Jacobi iterative
                    // solver.  The average normalized residual is initialized
                    // to a large value.
                    //double avg_norm_res;
                    double max_norm_res;

                    // Write a log file of the normalized residuals during the
                    // Jacobi iterations
                    std::ofstream reslog;
                    if (write_res_log == 1)
                        reslog.open("max_res_norm.dat");

                    for (unsigned int nijac = 0; nijac<darcy.maxiter; ++nijac) {

#if defined(REPORT_EPSILON) || defined(REPORT_FORCING_TERMS)
                std::cout << "\n\n### Jacobi iteration " << nijac << std::endl;
#endif

                        if (nijac == 0) {
                            if (PROFILING == 1)
                                startTimer(&kernel_tic);
                            copyValues<Float><<<dimGridFluid, dimBlockFluid>>>(
                                    dev_darcy_p,
                                    dev_darcy_p_old);
                            hipDeviceSynchronize();
                            if (PROFILING == 1)
                                stopTimer(&kernel_tic, &kernel_toc,
                                        &kernel_elapsed, &t_copyValues);
                            checkForCudaErrorsIter(
                                    "Post copyValues(p -> p_old)", iter);
                        }

                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        setDarcyGhostNodes<Float>
                            <<<dimGridFluid, dimBlockFluid>>>(
                                dev_darcy_p,
                                darcy.bc_xn, darcy.bc_xp,
                                darcy.bc_yn, darcy.bc_yp,
                                darcy.bc_bot, darcy.bc_top);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_setDarcyGhostNodes);
                        checkForCudaErrorsIter("Post setDarcyGhostNodes("
                                "dev_darcy_p) in Jacobi loop", iter);

                        if (nijac == 0) {
                            if (PROFILING == 1)
                                startTimer(&kernel_tic);
                            firstDarcySolution<<<dimGridFluid, dimBlockFluid>>>(
                                    dev_darcy_p,
                                    dev_darcy_k,
                                    dev_darcy_phi,
                                    dev_darcy_dphi,
                                    dev_darcy_div_v_p,
                                    dev_darcy_vp_avg,
                                    dev_darcy_grad_k,
                                    darcy.beta_f,
                                    darcy.mu,
                                    darcy.bc_xn,
                                    darcy.bc_xp,
                                    darcy.bc_yn,
                                    darcy.bc_yp,
                                    darcy.bc_bot,
                                    darcy.bc_top,
                                    darcy.ndem,
                                    wall0_iz,
                                    dev_darcy_p_constant,
                                    dev_darcy_dp_expl);
                            hipDeviceSynchronize();
                            if (PROFILING == 1)
                                stopTimer(&kernel_tic, &kernel_toc,
                                        &kernel_elapsed,
                                        &t_updateDarcySolution);
                            checkForCudaErrorsIter("Post updateDarcySolution",
                                    iter);
                        }

                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        updateDarcySolution<<<dimGridFluid, dimBlockFluid>>>(
                                dev_darcy_p_old,
                                //dev_darcy_dpdt,
                                dev_darcy_dp_expl,
                                dev_darcy_p,
                                dev_darcy_k,
                                dev_darcy_phi,
                                dev_darcy_dphi,
                                dev_darcy_div_v_p,
                                dev_darcy_vp_avg,
                                dev_darcy_grad_k,
                                darcy.beta_f,
                                darcy.mu,
                                darcy.bc_xn,
                                darcy.bc_xp,
                                darcy.bc_yn,
                                darcy.bc_yp,
                                darcy.bc_bot,
                                darcy.bc_top,
                                darcy.ndem,
                                wall0_iz,
                                dev_darcy_p_constant,
                                dev_darcy_p_new,
                                dev_darcy_norm);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_updateDarcySolution);
                        checkForCudaErrorsIter("Post updateDarcySolution",
                                iter);

                        if (darcy.bc_top == 1) {
                            if (PROFILING == 1)
                                startTimer(&kernel_tic);
                            setDarcyTopWallFixedFlow
                                <<<dimGridFluid, dimBlockFluid>>>
                                (wall0_iz, dev_darcy_p);
                            hipDeviceSynchronize();
                            if (PROFILING == 1)
                                stopTimer(&kernel_tic, &kernel_toc,
                                        &kernel_elapsed,
                                        &t_updateDarcySolution);
                            checkForCudaErrorsIter(
                                    "Post setDarcyTopWallFixedFlow", iter);
                        }

                        if (darcy.bc_bot == 4 || darcy.bc_top == 4) {
                            if (PROFILING == 1)
                                startTimer(&kernel_tic);
                            setDarcyGhostNodesFlux<Float>
                                <<<dimGridFluid, dimBlockFluid>>>(
                                        dev_darcy_p,
                                        darcy.bc_bot,
                                        darcy.bc_top,
                                        darcy.bc_bot_flux,
                                        darcy.bc_top_flux,
                                        dev_darcy_k,
                                        darcy.mu);
                            hipDeviceSynchronize();
                            if (PROFILING == 1)
                                stopTimer(&kernel_tic, &kernel_toc,
                                        &kernel_elapsed,
                                        &t_setDarcyGhostNodes);
                            checkForCudaErrorsIter(
                                    "Post setDarcyGhostNodesFlux", iter);
                        }

                        // Copy new values to current values
                        if (PROFILING == 1)
                            startTimer(&kernel_tic);
                        copyValues<Float><<<dimGridFluid, dimBlockFluid>>>(
                                dev_darcy_p_new,
                                dev_darcy_p);
                        hipDeviceSynchronize();
                        if (PROFILING == 1)
                            stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                    &t_copyValues);
                        checkForCudaErrorsIter("Post copyValues(p_new -> p)",
                                iter);

#ifdef REPORT_EPSILON
                        std::cout << "\n###### JACOBI ITERATION "
                            << nijac << " after copyValues ######" << std::endl;
                        transferDarcyPressuresFromGlobalDeviceMemory();
                        printDarcyArray(stdout, darcy.p, "p");
#endif

                        if (nijac % nijacnorm == 0) {
                            // Read the normalized residuals from the device
                            transferDarcyNormFromGlobalDeviceMemory();

                            // Write the normalized residuals to the terminal
                            //printDarcyArray(stdout, darcy.norm, "norm");

                            // Find the maximum value of the normalized
                            // residuals
                            max_norm_res = maxNormResDarcy();

                            // Write the Jacobi iteration number and maximum
                            // value of the normalized residual to the log file
                            if (write_res_log == 1)
                                reslog << nijac << '\t' << max_norm_res
                                    << std::endl;

                            if (max_norm_res <= darcy.tolerance) {
                                if (write_conv_log == 1
                                        && iter % conv_log_interval == 0)
                                    convlog << iter+1 << '\t' << nijac
                                        << std::endl;

                                break;  // solution has converged
                            }
                        }

                        if (nijac == darcy.maxiter-1) {

                            if (write_conv_log == 1)
                                convlog << iter+1 << '\t' << nijac << std::endl;

                            std::cerr << "\nIteration " << iter << ", time " 
                                << iter*time.dt << " s: "
                                "Error, the pressure solution in the fluid "
                                "calculations did not converge. Try increasing "
                                "the value of 'darcy.maxiter' ("
                                << darcy.maxiter
                                << ") or increase 'darcy.tolerance' ("
                                << darcy.tolerance << ")." << std::endl;
                        }

                        if (write_res_log == 1)
                            reslog.close();

                        //break; // end after first iteration
                    }

                    // Zero all dphi values right after they are used in fluid
                    // solution, unless a file is written in this step.
                    if (filetimeclock + time.dt < time.file_dt) {
                        setDarcyZeros<Float> <<<dimGridFluid, dimBlockFluid>>>
                            (dev_darcy_dphi);
                        hipDeviceSynchronize();
                        checkForCudaErrorsIter(
                                "After setDarcyZeros(dev_darcy_dphi)", iter);
                    }

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    setDarcyGhostNodes<Float> <<<dimGridFluid, dimBlockFluid>>>
                        (dev_darcy_p,
                         darcy.bc_xn, darcy.bc_xp,
                         darcy.bc_yn, darcy.bc_yp,
                         darcy.bc_bot, darcy.bc_top);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_setDarcyGhostNodes);
                    checkForCudaErrorsIter("Post setDarcyGhostNodes("
                            "dev_darcy_p) after Jacobi loop", iter);

                    if (PROFILING == 1)
                        startTimer(&kernel_tic);
                    findDarcyVelocities<<<dimGridFluid, dimBlockFluid>>>(
                            dev_darcy_p,
                            dev_darcy_phi,
                            dev_darcy_k,
                            darcy.mu,
                            dev_darcy_v);
                    hipDeviceSynchronize();
                    if (PROFILING == 1)
                        stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                                &t_findDarcyVelocities);
                    checkForCudaErrorsIter("Post findDarcyVelocities", iter);
                }
            }
        }
        //break; // end after first iteration

        if (np > 0) {

            // Find shear stresses on upper fixed particles if a shear stress BC
            // is specified (wmode[0] == 3)
            if (walls.nw > 0 && walls.wmode[0] == 3) {

                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                findShearStressOnFixedMovingParticles<<<dimGrid, dimBlock>>>
                    (dev_x,
                     dev_vel,
                     dev_force,
                     dev_walls_tau_eff_x_pp);
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_summation);
                checkForCudaErrorsIter(
                        "Post findShearStressOnFixedMovingParticles", iter);

                if (PROFILING == 1)
                    startTimer(&kernel_tic);
                summation<<<dimGrid, dimBlock>>>(dev_walls_tau_eff_x_pp,
                        dev_walls_tau_eff_x_partial);
                hipDeviceSynchronize();
                if (PROFILING == 1)
                    stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                            &t_summation);
                checkForCudaErrorsIter("Post shear stress summation", iter);
            }

            // Determine whether it is time to step the velocity
            if (time.current >= v2_start && time.current < v2_end &&
                    velocity_state == 1) {
                change_velocity_state = 1.0;
                velocity_state = 2;
            } else if (time.current >= v2_end && velocity_state == 2) {
                change_velocity_state = -1.0;
                velocity_state = 1;
            }

            // Update particle kinematics
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            integrate<<<dimGrid, dimBlock>>>(dev_x_sorted, 
                    dev_vel_sorted, 
                    dev_angvel_sorted,
                    dev_x, 
                    dev_vel, 
                    dev_angvel,
                    dev_force,
                    dev_torque, 
                    dev_angpos,
                    dev_acc,
                    dev_angacc,
                    dev_vel0,
                    dev_angvel0,
                    dev_xyzsum,
                    dev_gridParticleIndex,
                    iter,
                    dev_walls_wmode,
                    dev_walls_mvfd,
                    dev_walls_tau_eff_x_partial,
                    dev_walls_tau_x,
                    walls.tau_x[0],
                    change_velocity_state,
                    velocity_factor,
                    blocksPerGrid);
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Post integrate", iter);
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_integrate);

            if (change_velocity_state != 0)
                change_velocity_state = 0;

            // Summation of forces on wall
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            if (walls.nw > 0) {
                summation<<<dimGrid, dimBlock>>>(dev_walls_force_pp,
                        dev_walls_force_partial);
            }
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_summation);
            checkForCudaErrorsIter("Post wall force summation", iter);

            // Update wall kinematics
            if (PROFILING == 1)
                startTimer(&kernel_tic);
            if (walls.nw > 0) {
                integrateWalls<<< 1, walls.nw>>>(
                        dev_walls_nx,
                        dev_walls_mvfd,
                        dev_walls_wmode,
                        dev_walls_force_partial,
                        dev_walls_acc,
                        blocksPerGrid,
                        time.current,
                        iter);
            }
            hipDeviceSynchronize();
            if (PROFILING == 1)
                stopTimer(&kernel_tic, &kernel_toc, &kernel_elapsed,
                        &t_integrateWalls);
            checkForCudaErrorsIter("Post integrateWalls", iter);
        }

        // Update timers and counters
        //time.current  = iter*time.dt;
        time.current  += time.dt;
        filetimeclock += time.dt;
        ++iter;

        // Make sure all preceding tasks are complete
        if (hipDeviceSynchronize() != hipSuccess) {
            cerr << "Error during hipDeviceSynchronize()" << endl;
        }

        // Report time to console
        if (verbose == 1 && (iter % stdout_report == 0)) {

            toc = clock();
            time_spent = (toc - tic)/(CLOCKS_PER_SEC); // real time spent

            // Real time it takes to compute a second of model time
            t_ratio = time_spent/(time.current - t_start);
            time_t estimated_seconds_left(t_ratio*(time.total - time.current));
            tm *time_eta = gmtime(&estimated_seconds_left);

            cout << "\r  Current time: " << time.current << "/"
                << time.total << " s. ("
                << t_ratio << " s_real/s_sim, ETA: "
                << time_eta->tm_yday << "d "
                << std::setw(2) << std::setfill('0') << time_eta->tm_hour << ":"
                << std::setw(2) << std::setfill('0') << time_eta->tm_min << ":"
                << std::setw(2) << std::setfill('0') << time_eta->tm_sec
                << ")       "; // << std::flush;
        }


        // Produce output binary if the time interval 
        // between output files has been reached
        if (filetimeclock >= time.file_dt) {

            // Pause the CPU thread until all CUDA calls previously issued are
            // completed
            hipDeviceSynchronize();
            checkForCudaErrorsIter("Beginning of file output section", iter);

            // v_x, v_y, v_z -> v
            if (fluid == 1 && cfd_solver == 0) {
                interpolateFaceToCenter<<<dimGridFluid, dimBlockFluid>>>(
                        dev_ns_v_x,
                        dev_ns_v_y,
                        dev_ns_v_z,
                        dev_ns_v);
                hipDeviceSynchronize();
                checkForCudaErrorsIter("Post interpolateFaceToCenter", iter);
            }

            //// Copy device data to host memory
            transferFromGlobalDeviceMemory();
            checkForCudaErrorsIter("After transferFromGlobalDeviceMemory()",
                    iter);

            // Empty the dphi values after device to host transfer
            if (fluid == 1) {
                if (cfd_solver == 1) {
                    setDarcyZeros<Float> <<<dimGridFluid, dimBlockFluid>>>
                        (dev_darcy_dphi);
                    hipDeviceSynchronize();
                    checkForCudaErrorsIter(
                            "After setDarcyZeros(dev_darcy_dphi) after transfer",
                            iter);
                }
            }

            // Pause the CPU thread until all CUDA calls previously issued are
            // completed
            hipDeviceSynchronize();

            // Check the numerical stability of the NS solver
            if (fluid == 1)
                if (cfd_solver == 0)
                    checkNSstability();

            // Write binary output file
            time.step_count += 1;
            snprintf(file, sizeof(file), "output/%s.output%05d.bin",
                     sid.c_str(), time.step_count);
            writebin(file);

            /*std::cout
              << "\n###### OUTPUT FILE " << time.step_count << " ######"
                << std::endl;
            transferNSepsilonFromGlobalDeviceMemory();
            printNSarray(stdout, ns.epsilon, "epsilon");*/

            // Write fluid arrays
            /*if (fluid == 1 && cfd_solver == 0) {
                sprintf(file,"output/%s.ns_phi.output%05d.bin", sid.c_str(),
                    time.step_count);
                writeNSarray(ns.phi, file);
            }*/

            if (CONTACTINFO == 1) {
                // Write contact information to stdout
                cout << "\n\n---------------------------\n"
                    << "t = " << time.current << " s.\n"
                    << "---------------------------\n";

                for (int n = 0; n < np; ++n) {
                    cout << "\n## Particle " << n << " ##\n";

                    cout  << "- contacts:\n";
                    for (int nc = 0; nc < NC; ++nc) 
                        cout << "[" << nc << "]=" << k.contacts[nc+NC*n] <<
                            '\n';

                    cout << "\n- delta_t:\n";
                    for (int nc = 0; nc < NC; ++nc) 
                        cout << k.delta_t[nc+NC*n].x << '\t'
                            << k.delta_t[nc+NC*n].y << '\t'
                            << k.delta_t[nc+NC*n].z << '\t'
                            << k.delta_t[nc+NC*n].w << '\n';

                    cout << "\n- distmod:\n";
                    for (int nc = 0; nc < NC; ++nc) 
                        cout << k.distmod[nc+NC*n].x << '\t'
                            << k.distmod[nc+NC*n].y << '\t'
                            << k.distmod[nc+NC*n].z << '\t'
                            << k.distmod[nc+NC*n].w << '\n';
                }
                cout << '\n';
            }

            // Update status.dat at the interval of filetime 
            outfile = "output/" + sid + ".status.dat";
            fp = fopen(outfile.c_str(), "w");
            fprintf(fp,"%2.4e %2.4e %d\n", 
                    time.current, 
                    100.0*time.current/time.total,
                    time.step_count);
            fclose(fp);

            filetimeclock = 0.0;
        }

        // Uncomment break command to stop after the first iteration
        //break;
    }

    if (write_conv_log == 1)
        convlog.close();


    // Stop clock and display calculation time spent
    toc = clock();
    hipEventRecord(dev_toc, 0);
    hipEventSynchronize(dev_toc);

    time_spent = (toc - tic)/(CLOCKS_PER_SEC);
    hipEventElapsedTime(&dev_time_spent, dev_tic, dev_toc);

    if (verbose == 1) {
        cout << "\nSimulation ended. Statistics:\n"
            << "  - Last output file number: " 
            << time.step_count << "\n"
            << "  - GPU time spent: "
            << dev_time_spent/1000.0f << " s\n"
            << "  - CPU time spent: "
            << time_spent << " s\n"
            << "  - Mean duration of iteration:\n"
            << "      " << dev_time_spent/((double)iter*1000.0f) << " s"
            << std::endl; 
    }

    hipEventDestroy(dev_tic);
    hipEventDestroy(dev_toc);

    hipEventDestroy(kernel_tic);
    hipEventDestroy(kernel_toc);

    // Report time spent on each kernel
    if (PROFILING == 1 && verbose == 1) {
        double t_sum = t_calcParticleCellID + t_thrustsort + t_reorderArrays +
            t_topology + t_interact + t_bondsLinear + t_latticeBoltzmannD3Q19 +
            t_integrate + t_summation + t_integrateWalls + t_findPorositiesDev +
            t_findNSstressTensor +
            t_findNSdivphiviv + t_findNSdivtau + t_findPredNSvelocities +
            t_setNSepsilon + t_setNSdirichlet + t_setNSghostNodesDev +
            t_findNSforcing + t_jacobiIterationNS + t_updateNSvelocityPressure +
            t_findDarcyPorosities + t_setDarcyGhostNodes +
            t_findDarcyPressureForce + t_setDarcyTopPressure +
            t_findDarcyPermeabilities + t_findDarcyPermeabilityGradients +
            //t_findDarcyPressureChange +
            t_updateDarcySolution + t_copyValues + t_findDarcyVelocities;

        cout << "\nKernel profiling statistics:\n"
            << "  - calcParticleCellID:\t\t" << t_calcParticleCellID/1000.0
            << " s"
            << "\t(" << 100.0*t_calcParticleCellID/t_sum << " %)\n"
            << "  - thrustsort:\t\t\t" << t_thrustsort/1000.0 << " s"
            << "\t(" << 100.0*t_thrustsort/t_sum << " %)\n"
            << "  - reorderArrays:\t\t" << t_reorderArrays/1000.0 << " s"
            << "\t(" << 100.0*t_reorderArrays/t_sum << " %)\n";
        if (params.contactmodel == 2 || params.contactmodel == 3) {
            cout
            << "  - topology:\t\t\t" << t_topology/1000.0 << " s"
            << "\t(" << 100.0*t_topology/t_sum << " %)\n";
        }
        cout << "  - interact:\t\t\t" << t_interact/1000.0 << " s"
            << "\t(" << 100.0*t_interact/t_sum << " %)\n";
        if (params.nb0 > 0) {
            cout << "  - bondsLinear:\t\t" << t_bondsLinear/1000.0 << " s"
            << "\t(" << 100.0*t_bondsLinear/t_sum << " %)\n";
        }
        cout << "  - integrate:\t\t\t" << t_integrate/1000.0 << " s"
            << "\t(" << 100.0*t_integrate/t_sum << " %)\n"
            << "  - summation:\t\t\t" << t_summation/1000.0 << " s"
            << "\t(" << 100.0*t_summation/t_sum << " %)\n"
            << "  - integrateWalls:\t\t" << t_integrateWalls/1000.0 << " s"
            << "\t(" << 100.0*t_integrateWalls/t_sum << " %)\n";
        if (fluid == 1 && cfd_solver == 0) {
            cout << "  - findPorositiesDev:\t\t" << t_findPorositiesDev/1000.0
                << " s" << "\t(" << 100.0*t_findPorositiesDev/t_sum << " %)\n"
                << "  - findNSstressTensor:\t\t" << t_findNSstressTensor/1000.0
                << " s" << "\t(" << 100.0*t_findNSstressTensor/t_sum << " %)\n"
                << "  - findNSdivphiviv:\t\t" << t_findNSdivphiviv/1000.0
                << " s" << "\t(" << 100.0*t_findNSdivphiviv/t_sum << " %)\n"
                << "  - findNSdivtau:\t\t" << t_findNSdivtau/1000.0
                << " s" << "\t(" << 100.0*t_findNSdivtau/t_sum << " %)\n"
                << "  - findPredNSvelocities:\t" <<
                t_findPredNSvelocities/1000.0 << " s" << "\t(" <<
                100.0*t_findPredNSvelocities/t_sum << " %)\n"
                << "  - setNSepsilon:\t\t" << t_setNSepsilon/1000.0
                << " s" << "\t(" << 100.0*t_setNSepsilon/t_sum << " %)\n"
                << "  - setNSdirichlet:\t\t" << t_setNSdirichlet/1000.0
                << " s" << "\t(" << 100.0*t_setNSdirichlet/t_sum << " %)\n"
                << "  - setNSghostNodesDev:\t\t" << t_setNSghostNodesDev/1000.0
                << " s" << "\t(" << 100.0*t_setNSghostNodesDev/t_sum << " %)\n"
                << "  - findNSforcing:\t\t" << t_findNSforcing/1000.0 << " s"
                << "\t(" << 100.0*t_findNSforcing/t_sum << " %)\n"
                << "  - jacobiIterationNS:\t\t" << t_jacobiIterationNS/1000.0
                << " s"
                << "\t(" << 100.0*t_jacobiIterationNS/t_sum << " %)\n"
                << "  - updateNSvelocityPressure:\t"
                << t_updateNSvelocityPressure/1000.0 << " s"
                << "\t(" << 100.0*t_updateNSvelocityPressure/t_sum << " %)\n";
        } else if (fluid == 1 && cfd_solver == 1) {
            cout << "  - findDarcyPorosities:\t" <<
                t_findDarcyPorosities/1000.0 << " s" << "\t(" <<
                100.0*t_findDarcyPorosities/t_sum << " %)\n"
                << "  - setDarcyGhostNodes:\t\t" <<
                t_setDarcyGhostNodes/1000.0 << " s" << "\t(" <<
                100.0*t_setDarcyGhostNodes/t_sum << " %)\n"
                << "  - findDarcyPressureForce:\t" <<
                t_findDarcyPressureForce/1000.0 << " s" << "\t(" <<
                100.0*t_findDarcyPressureForce/t_sum << " %)\n"
                << "  - setDarcyTopPressure:\t" <<
                t_setDarcyTopPressure/1000.0 << " s" << "\t(" <<
                100.0*t_setDarcyTopPressure/t_sum << " %)\n"
                << "  - findDarcyPermeabilities:\t" <<
                t_findDarcyPermeabilities/1000.0 << " s" << "\t(" <<
                100.0*t_findDarcyPermeabilities/t_sum << " %)\n"
                << "  - findDarcyPermeabilityGrads:\t" <<
                t_findDarcyPermeabilityGradients/1000.0 << " s" << "\t(" <<
                100.0*t_findDarcyPermeabilityGradients/t_sum << " %)\n"
                //<< "  - findDarcyPressureChange:\t" <<
                //t_findDarcyPressureChange/1000.0 << " s" << "\t(" <<
                //100.0*t_findDarcyPressureChange/t_sum << " %)\n"
                << "  - updateDarcySolution:\t" <<
                t_updateDarcySolution/1000.0 << " s" << "\t(" <<
                100.0*t_updateDarcySolution/t_sum << " %)\n"
                << "  - copyValues:\t\t\t" <<
                t_copyValues/1000.0 << " s" << "\t(" <<
                100.0*t_copyValues/t_sum << " %)\n"
                << "  - findDarcyVelocities:\t" <<
                t_findDarcyVelocities/1000.0 << " s" << "\t(" <<
                100.0*t_findDarcyVelocities/t_sum << " %)" << std::endl;
        }
    }

    // Free GPU device memory
    freeGlobalDeviceMemory();
    checkForCudaErrorsIter("After freeGlobalDeviceMemory()", iter);

    // Free contact info arrays
    delete[] k.contacts;
    delete[] k.distmod;
    delete[] k.delta_t;

    if (fluid == 1) {
        if (cfd_solver == 0)
            endNS();
        else if (cfd_solver == 1)
            endDarcy();
    }

    hipDeviceReset();
}
// vim: tabstop=8 expandtab shiftwidth=4 softtabstop=4
